#include <hip/hip_runtime.h>
#include <cmath>

// Structure to hold information about a single term in a document
struct TermInfo {
    int term_id;
    float tf;
};

// The CUDA kernel that runs on the GPU.
// Each thread calculates the BM25 score for one document.
__global__ void bm25_kernel(
    const TermInfo* docs_terms,      // Flattened array of all terms in all documents
    const int* doc_offsets,          // Start index for each document in docs_terms
    const int* doc_lengths,          // Length of each document (in words)
    const int* query_term_ids,       // Array of term IDs in the query
    const float* idf_scores,         // IDF score for each query term
    int num_docs,                    // Total number of documents in this batch
    int num_query_terms,             // Number of terms in the query
    float avg_doc_length,            // Average document length for the whole corpus
    float k1,                        // BM25 k1 parameter
    float b,                         // BM25 b parameter
    float* out_scores                // Output array to store the scores
) {
    // Get the unique ID for this thread, which corresponds to the document index
    int doc_idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread ID is within the bounds of our document batch
    if (doc_idx < num_docs) {
        float total_score = 0.0f;
        
        // Get the start and end offsets for the current document's terms
        int start_offset = doc_offsets[doc_idx];
        int end_offset = (doc_idx == num_docs - 1) ? doc_offsets[doc_idx + 1] : doc_offsets[doc_idx + 1]; // Special handling for last doc
                                                                                                        // This is a placeholder, a better way is to pass the size of docs_terms
        if (doc_idx == num_docs - 1) {
            // A more robust way to get the end offset would be needed if not passed explicitly.
            // For this example, we'll assume it's handled by the calling Python code.
            // A simpler approach might be to also pass doc_term_counts array.
        }


        // Loop over each term in the search query
        for (int i = 0; i < num_query_terms; ++i) {
            int query_term_id = query_term_ids[i];
            float term_idf = idf_scores[i];
            float term_tf = 0.0f;

            // Find the term frequency (TF) of the query term in the current document
            for (int j = start_offset; j < end_offset; ++j) {
                if (docs_terms[j].term_id == query_term_id) {
                    term_tf = docs_terms[j].tf;
                    break; // Found the term, no need to search further
                }
            }

            // If the term exists in the document, calculate its BM25 contribution
            if (term_tf > 0.0f) {
                float doc_len = (float)doc_lengths[doc_idx];
                
                // BM25 formula for this term
                float numerator = term_idf * term_tf * (k1 + 1.0f);
                float denominator = term_tf + k1 * (1.0f - b + b * (doc_len / avg_doc_length));
                total_score += numerator / denominator;
            }
        }
        // Write the final score for this document to the output array
        out_scores[doc_idx] = total_score;
    }
}
